
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__
void add(int n, float* x, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// int stride = blockDim.x * gridDim.x;

	if (index < n) {
		y[index] = x[index] + y[index];
	}
	// for (int i = index; i < n; i += stride) {
	// 	y[i] = x[i] + y[i];
	// }	
}


int main() {
	int N = 1 << 28;
	float *x, *y;

	hipMallocManaged(&x, N * sizeof(float));
	hipMallocManaged(&y, N * sizeof(float));


	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;

	int numBlocks = (N + blockSize - 1) / blockSize;

	add<<<numBlocks, blockSize>>>(N, x, y);

	hipDeviceSynchronize();	
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	hipFree(x);
	hipFree(y);
	return 0;
}
